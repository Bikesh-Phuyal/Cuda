#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
This program shows a poor way to implement a password cracker in CUDA C. It's poor because it accomplishes this with a single thread, which definitely is not good given the scale to which CUDA programs have parallelism.
The intentions of this program are:
1) Demonstrate the use of __device__ and __global__ functions
2) Enable a simulation of password cracking in the absence of library
with equivalent functionality to libcrypt. The password to be found
is hardcoded into a function called is_a_match.
Compile and run with:
nvcc -o passwordcrack_cuda passwordcrack_cuda.cu
./passwordcrack_cuda
Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/
/****************************************************************************
This function returns 1 if the attempt at cracking the password is
identical to the plain text password string stored in the program.
Otherwise,it returns 0.
*****************************************************************************/
__device__ int is_a_match(char *attempt) {
char plain_password1[] = "BP9843";
char plain_password2[] = "RP6870";
char plain_password3[] = "AP6498";
char plain_password4[] = "IP1354";
char *m = attempt;
char *s = attempt;
char *c = attempt;
char *d = attempt;
char *p1 = plain_password1;
char *p2 = plain_password2;
char *p3 = plain_password3;
char *p4 = plain_password4;
while(*m == *p1) {
if(*m == '\0')
{
printf("Password: %s\n",plain_password1);
break;
}
m++;
p1++;
}while(*s == *p2) {
if(*s == '\0')
{
printf("Password: %s\n",plain_password2);
break;
}
s++;
p2++;
}
while(*c == *p3) {
if(*c == '\0')
{
printf("Password: %s\n",plain_password3);
break;
}
c++;
p3++;
}
while(*d == *p4) {
if(*d == '\0')
{
printf("Password: %s\n",plain_password4);
return 1;
}
d++;
p4++;
}
return 0;
}
__global__ void kernel() {
char i1,i2,i3,i4; //variables
char password[7];
password[6] = '\0';
//block id thread id initilized
int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstMatch = i;
char secondMatch = j;
password[0] = firstMatch;
password[1] = secondMatch;
for(i1='0'; i1<='9'; i1++){
for(i2='0'; i2<='9'; i2++){
for(i3='0'; i3<='9'; i3++){
for(i4='0'; i4<='9'; i4++){
password[2] = i1;
password[3] = i2;
password[4] = i3;
password[5] = i4;
if(is_a_match(password)) {
}
else {
//printf("tried: %s\n", password);
}}
}
}
}
}
//time difference
int time_difference(struct timespec *start,
struct timespec *finish,
long long int *difference) {
long long int ds = finish->tv_sec - start->tv_sec;
long long int dn = finish->tv_nsec - start->tv_nsec;
if(dn < 0 ) {
ds--;
dn += 1000000000;
}
*difference = ds * 1000000000 + dn;
return !(*difference > 0);
}
int main() {
struct timespec start, finish;
long long int time_elapsed;
clock_gettime(CLOCK_MONOTONIC, &start);
// Kernel function to evaluate the block and thread to be used
 kernel <<<26,26>>>();
hipDeviceSynchronize();
clock_gettime(CLOCK_MONOTONIC, &finish);
time_difference(&start, &finish, &time_elapsed);
printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
return 0;
}
